#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/core.hpp>

__global__ void dummyKernel() {}

void RunDummyCudaKernel() {
    dummyKernel << <1, 1 >> > ();
    hipDeviceSynchronize();
}


__global__ void sobelKernel(const uchar* input, uchar* output, int width, int height, int step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x <= 0 || x >= width - 1 || y <= 0 || y >= height - 1) return;

    int gx = 0;
    int gy = 0;

    // grayscale from BGR
    auto getGray = [&](int xx, int yy) -> int {
        int offset = yy * step + xx * 3;
        return (int)(0.299f * input[offset + 2] + 0.587f * input[offset + 1] + 0.114f * input[offset]);
        };

    // Sobel X and Y kernels
    int sx[3][3] = { {-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1} };
    int sy[3][3] = { { 1,  2,  1}, {0, 0, 0}, {-1, -2, -1} };

    for (int j = -1; j <= 1; ++j)
        for (int i = -1; i <= 1; ++i) {
            int gray = getGray(x + i, y + j);
            gx += gray * sx[j + 1][i + 1];
            gy += gray * sy[j + 1][i + 1];
        }

    int mag = min(255, abs(gx) + abs(gy));

    int offset = y * step + x * 3;
    output[offset + 0] = mag; // B
    output[offset + 1] = mag; // G
    output[offset + 2] = mag; // R
}


__global__ void blurKernel(const uchar* input, uchar* output, int width, int height, int step)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
    {
        for (int c = 0; c < 3; ++c)
        {
            int sum = 0;
            for (int j = -1; j <= 1; ++j)
                for (int i = -1; i <= 1; ++i)
                {
                    int idx = (y + j) * step + (x + i) * 3 + c;
                    sum += input[idx];
                }

            int outIdx = y * step + x * 3 + c;
            output[outIdx] = sum / 9;
        }
    }
}



__global__ void invertKernel(const uchar* input, uchar* output, int width, int height, int step) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    // uchar pointer offset using row stride (step)
    const uchar* row_in = input + y * step;
    uchar* row_out = output + y * step;

    row_out[x] = 255 - row_in[x];
}


cv::Mat ApplyCudaFilterEdge(const cv::Mat& input)
{
    cv::Mat output(input.size(), input.type());

    uchar* d_input = nullptr;
    uchar* d_output = nullptr;
    size_t size = input.step * input.rows;

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, input.data, size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((input.cols + 15) / 16, (input.rows + 15) / 16);
    sobelKernel<<<grid, block>>>(d_input, d_output, input.cols, input.rows, input.step);
    hipDeviceSynchronize();

    hipMemcpy(output.data, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}


cv::Mat ApplyCudaInvert(const cv::Mat& input) 
{
    int width = input.cols;
    int height = input.rows;
    int step = input.step;

    uchar* d_input = nullptr, * d_output = nullptr;
    cv::Mat output(input.size(), input.type());

    size_t dataSize = height * step;

    // Allocate GPU memory
    hipMalloc(&d_input, dataSize);
    hipMalloc(&d_output, dataSize);

    // Copy input data to device
    hipMemcpy(d_input, input.data, dataSize, hipMemcpyHostToDevice);

    // Launch CUDA kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + 15) / 16, (height + 15) / 16);

    invertKernel << <gridSize, blockSize >> > (d_input, d_output, width, height, step);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(output.data, d_output, dataSize, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);

    return output;
}


cv::Mat ApplyCudaBlur(const cv::Mat& input)
{
    cv::Mat output(input.size(), input.type());

    uchar* d_input = nullptr;
    uchar* d_output = nullptr;
    size_t size = input.step * input.rows;

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, input.data, size, hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((input.cols + 15) / 16, (input.rows + 15) / 16);

    blurKernel << <grid, block >> > (d_input, d_output, input.cols, input.rows, input.step);
    hipDeviceSynchronize();

    hipMemcpy(output.data, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}








